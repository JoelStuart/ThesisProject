#include "hip/hip_runtime.h"
//Compile method within MATLAB -> | nvcc -ptx -arch=sm_21 Prop_demo.cu'
 
#include "sgp4unit.cu"
 
//This function 
__global__ void PartPd_Kernel(double* ResultOut, int rows, int cols, int t,
	    double del, double* jdsatepoch, double* bstar,
        double* ecco, double* argpo, double* inclo, double* mo, double* no,
        double* nodeo, double* tse)
	{
    //Determine Thread Position in Grid
    double rowD = blockIdx.y * blockDim.y + threadIdx.y;
    double colD = blockIdx.x * blockDim.x + threadIdx.x;
	int row = (int) rowD;
	int col = (int) colD;
	gravconsttype whichconst = wgs72old;
	elsetrec satrec;
	
	
    
    //Ensure Excess Threads are not Evaluated
    if ((row < rows)&&(col < cols)){
		sgp4init(whichconst, 'i',  1,  jdsatepoch[col], bstar[col],
                  ecco[col],  argpo[col],  inclo[col],  mo[col],  no[col],
                  nodeo[col], satrec);
		double time = (tse[col]*1440) + ((t + rowD)*del);
		double ro[3];
		double vo[3];
		
		sgp4(whichconst, satrec, time, ro, vo);

        ResultOut[row + rows*(col+cols*0)] = ro[0];
		ResultOut[row + rows*(col+cols*1)] = ro[1];
		ResultOut[row + rows*(col+cols*2)] = ro[2];
    }
	
}


